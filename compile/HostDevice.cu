#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include "MyDevice.h"

__global__ void frontEnd(){
  MyDevice md;
  md.print();
}

void HostDevice(unsigned int gd, unsigned int bd){
  frontEnd<<<gd, bd>>>();
  hipDeviceSynchronize();
}
