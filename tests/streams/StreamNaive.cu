#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

#include <omp.h>

#define N 262144
#include "macros.h"
#include "StreamNaive.cuh"

__global__ void kernel()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
    printf("Kernel on the default stream.\n");
}

__global__ void kernel_1(int stream)
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
    printf("Kernel 1 on stream %d.\n", stream);
}

__global__ void kernel_2(int stream)
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
    printf("Kernel 2 on stream %d.\n", stream);
}

__global__ void kernel_3(int stream)
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
    printf("Kernel 3 on stream %d.\n", stream);
}

__global__ void kernel_4(int stream)
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
    printf("Kernel 4 on stream %d.\n", stream);
}

StreamNaive::StreamNaive(int n){
  _n = n;
  //_streams = new cudaSream_t[_n];
  _streams = (hipStream_t *) malloc(_n * sizeof(hipStream_t));
  for(int i=0;i<_n;i++){
    CUDA_STATUS(hipStreamCreate(&(_streams[i])));
  }

  CUDA_STATUS(hipEventCreate(&_start));
  CUDA_STATUS(hipEventCreate(&_stop));
}

StreamNaive::~StreamNaive(){
  for(int i=0;i<_n;i++)
    CUDA_STATUS(hipStreamDestroy(_streams[i]));  
  delete [] _streams;

  CUDA_STATUS(hipEventDestroy(_start));
  CUDA_STATUS(hipEventDestroy(_stop));

  CUDA_STATUS(hipDeviceReset());
}

void StreamNaive::run(){
  dim3 block(1), grid(1);
  for (int i = 0; i < _n; i++){
      kernel_1<<<grid, block, 0, _streams[i]>>>(i);
      kernel_2<<<grid, block, 0, _streams[i]>>>(i);
      kernel_3<<<grid, block, 0, _streams[i]>>>(i);
      kernel_4<<<grid, block, 0, _streams[i]>>>(i);
  }
}

void StreamNaive::runBlock(){
  dim3 block(1), grid(1);

  int i = 0;
  kernel_1<<<grid, block, 0, _streams[i]>>>(i);
  kernel_2<<<grid, block, 0, _streams[i]>>>(i);
  kernel_3<<<grid, block, 0, _streams[i]>>>(i);
  kernel_4<<<grid, block, 0, _streams[i]>>>(i);

  kernel<<<grid, block>>>();
  
  for (i = 1; i < _n; i++){
      kernel_1<<<grid, block, 0, _streams[i]>>>(i);
      kernel_2<<<grid, block, 0, _streams[i]>>>(i);
      kernel_3<<<grid, block, 0, _streams[i]>>>(i);
      kernel_4<<<grid, block, 0, _streams[i]>>>(i);
  }
}

void StreamNaive::runomp(){
  dim3 block(1), grid(1);

  omp_set_num_threads(_n);
#pragma omp parallel
  {
    int i = omp_get_thread_num();
    kernel_1<<<grid, block, 0, _streams[i]>>>(i);
    kernel_2<<<grid, block, 0, _streams[i]>>>(i);
    kernel_3<<<grid, block, 0, _streams[i]>>>(i);
    kernel_4<<<grid, block, 0, _streams[i]>>>(i);
    cout << "Stream " << i << "has been launched!" << endl;
  }
}

void StreamNaive::setSync(_sync sync){
  switch(sync){
  case deviceSync:
    CUDA_STATUS(hipDeviceSynchronize());
    break;
  case streamSync:
    for (int i = 0; i < _n; i++){
      CUDA_STATUS(hipStreamSynchronize(_streams[i]));
    }
  }
}

void StreamNaive::start(){
  CUDA_STATUS(hipEventRecord(_start));
}

float StreamNaive::stop(){
  float dt;
  CUDA_STATUS(hipEventRecord(_stop));
  CUDA_STATUS(hipEventSynchronize(_stop));
  CUDA_STATUS(hipEventElapsedTime(&dt, _start, _stop));
  return dt;
}
