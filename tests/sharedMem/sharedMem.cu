#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <macros.h>
#include <CompTime.h>

#define DIMGRD 512
#define DIMBLK 1024

__device__ __host__ float fab(float a, float b){
  return a*b*a*b;
}

__global__ void sum_global(float *a_d, float *b_d, float *out_d){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  out_d[idx] = fab(a_d[idx],  b_d[idx]);
}

int main(){
  int dim = DIMBLK*DIMGRD;
    float *a_d, *b_d, *out_d;
  float a_h[dim], b_h[dim], out_h[dim];
  for(int i=0; i<dim; i++){
    a_h[i] = (float) i; b_h[i] = (float) i;
  }
  //cout << "a_h and b_h has just been initialized!" << endl;
  
  CompTime stopwatch;
  for(int i=0; i<dim; i++){
    out_h[i] = fab(a_h[i], b_h[i]);
  }

  cout << "CPU computation time: " << scientific << stopwatch.getTime() << " s.\n" << endl;
    
  //cout << "a_h and b_h has just been initialized!" << endl;

  size_t _nbytes = dim * sizeof(float);
  CUDA_STATUS(hipMalloc((void**) &a_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &b_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &out_d, _nbytes));
  CUDA_STATUS(hipMemcpy(a_d, a_h, _nbytes, hipMemcpyHostToDevice));
  CUDA_STATUS(hipMemcpy(b_d, b_h, _nbytes, hipMemcpyHostToDevice));
  stopwatch.reset();
  sum_global<<<DIMGRD, DIMBLK>>>(a_d, b_d, out_d);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess){
    printf("Kernel call in main:\n");
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(err), __LINE__, __FILE__);
    exit(1);
  }
  cout << "GPU with globalMem computation time with (" << DIMGRD << ", " << DIMBLK << "): " << stopwatch.getTime() << " s.\n" << endl;


  hipFree((void*) a_d);
  hipFree((void*) b_d);
  hipFree((void*) out_d);
  return 0;
}
