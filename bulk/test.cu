#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "fThread.cuh"

#define N 100


#define CUDA_STATUS(value){\
  hipError_t _m_cudaStat = value;\
  if ( _m_cudaStat != hipSuccess ){\
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
  }\
}

__global__ void fGrid(float* fIn_d, float *fOut_d, float t, float dt){
  fThread fi = fThread(fIn_d, fOut_d, t, dt);
  fi.print();
  fi.nextTime();
  fi.update();
}

int main(){
  int _ntot = N;
  int _nbytes = _ntot*sizeof(int);

  float f0_h[N], fOut_h[N];
  float *_fIn_d, *_fOut_d;
  float _t = 0.1, _dt = 0.1;
  for(int i=0;i<N;i++){
    f0_h[i] = (float) i;
  }
  
  CUDA_STATUS(hipMalloc((void**) &_fIn_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &_fOut_d, _nbytes));
  CUDA_STATUS(hipMemcpy(_fIn_d, f0_h, _nbytes, hipMemcpyHostToDevice));
  
  fGrid<<<1, _ntot>>>(_fIn_d, _fOut_d, _t, _dt);
  CUDA_STATUS(hipDeviceSynchronize());

  CUDA_STATUS(hipMemcpy(fOut_h, _fOut_d, _nbytes, hipMemcpyDeviceToHost));
  cout << "In output after hipMemcpy: " << endl;
  for(int i=0;i<=5;i++)
    cout << fOut_h[i] << endl;

  
  hipFree((void*) _fIn_d);
  hipFree((void*) _fOut_d);


  return 0;
}
