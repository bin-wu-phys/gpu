#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <cmath>
#include "InitCond.cuh"
#include "macros.h"

InitCond::InitCond(float tInit, Lattice* latt, float n0){
  _tInit = tInit; _latt = latt; _n0 = 0.5*n0/(PI*PI);
  _f0 = new float[_latt->get_nr()*_latt->get_nphit()*_latt->get_npt()*_latt->get_nvzt()];

  calc();
}

InitCond::~InitCond(){
  delete [] _f0;
}

float InitCond::Fbg(float r, float phir){
  return expf(-r*r);
}

float InitCond::Fp(float p){
  return 2.7305088f/(expf((p-1.0)/0.1) + 1.0);
}

float InitCond::Fv(float v){
  return 0.5;
}

void InitCond::calc(){
  int idx0, idx1, idx2, idx;
  for(int ir=0; ir<_latt->get_nr(); ir++){
    idx0 = ir;
    for(int iphi=0; iphi<_latt->get_nphit(); iphi++){
      idx1 = _latt->get_nphit()*idx0 + iphi;
      for(int ip=0; ip<_latt->get_npt(); ip++){
	idx2 = _latt->get_npt()*idx1 + ip;
	for(int iv=0; iv<_latt->get_nvzt(); iv++){
	  idx = _latt->get_nvzt()*idx2 + iv;
	  _f0[idx] = _n0*Fbg(_latt->get_r(ir), _latt->get_phit(iphi))*Fp(_latt->get_pt(ip))*Fv(_latt->get_vzt(iv));
	}
      }
    }
  }
}

void InitCond::toGlobalMem(float* f0_d){
  size_t nBytes = _latt->get_nr()*_latt->get_nphit()*_latt->get_npt()*_latt->get_nvzt()*sizeof(float);
  CUDA_STATUS(hipMemcpy(f0_d, _f0, nBytes, hipMemcpyHostToDevice));
}

float InitCond::get_tInit(){
  return _tInit;
}
