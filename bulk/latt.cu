#include "hip/hip_runtime.h"
#include "Lattice.cu"
#define NR 32
#define NPHI 21
#define NP 16
#define NV 64

__constant__ float rc[NR], phic[NPHI], pc[NP], vc[NV];

__global__ void read(){

  printf("r = ");
  for(int i=0; i<NR; i++){
    printf("%f ", rc[i]);
  }

  printf("\n\n");

  printf("phi = ");
  for(int i=0; i<NPHI; i++){
    printf("%f ", phic[i]);
  }

  printf("\n\n");

  printf("p = ");
  for(int i=0; i<NP; i++){
    printf("%f ", pc[i]);
  }

  printf("\n\n");

  printf("vz = ");
  for(int i=0; i<NV; i++){
    printf("%f ", vc[i]);
  }
}

int main(){
  Lattice latt(NR, NPHI, NP, NV, 3.0f, 3.0f, 15.0f);
  latt.toConstMem(rc, phic, pc, vc);

  read<<<1,1>>>();
  CUDA_STATUS(hipDeviceReset()); 
  return 0;
}
