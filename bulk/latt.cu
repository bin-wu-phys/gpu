#include "hip/hip_runtime.h"
/*This is used to test class Lattice.*/
#include "Lattice.cu"
#define NR 32
#define NPHI 21
#define NP 16
#define NV 64

__constant__ float rc[NR], phic[NPHI], pc[NP], vc[NV];

__global__ void read(){

  printf("r = ");
  for(int i=0; i<NR; i++){
    printf("%f ", rc[i]);
  }

  printf("\n\n");

  printf("phi = ");
  for(int i=0; i<NPHI; i++){
    printf("%f ", phic[i]);
  }

  printf("\n\n");

  printf("p = ");
  for(int i=0; i<NP; i++){
    printf("%f ", pc[i]);
  }

  printf("\n\n");

  printf("vz = ");
  for(int i=0; i<NV; i++){
    printf("%f ", vc[i]);
  }
}

int main(){
  Lattice latt(NR, NPHI, NP, NV, 3.0f, 3.0f, 15.0f);
  latt.toConstMem(rc, phic, pc, vc);

  cout << "r = ";
  for(int i=0; i< NR; i++){
    cout << latt.get_r(i) << " ";
  }
  cout << endl;

  cout << "phi = ";
  for(int i=0; i< NPHI; i++){
    cout << latt.get_phit(i) << " ";
  }
  cout << endl;
  
  cout << "p = ";
  for(int i=0; i< NP; i++){
    cout << latt.get_pt(i) << " ";
  }
  cout << endl;
  
  cout << "vz = ";
  for(int i=0; i< NV; i++){
    cout << latt.get_vzt(i) << " ";
  }
  cout << endl;

  read<<<1,1>>>();
  CUDA_STATUS(hipDeviceReset()); 
  return 0;
}
