#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "bulk.cuh"
#include "fThread.cuh"

#define CUDA_STATUS(value){\
  hipError_t _m_cudaStat = value;\
  if ( _m_cudaStat != hipSuccess ){\
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
  }\
}

__global__ void fGrid(float* fIn_d, float *fOut_d, float t, float dt){
  fThread fi = fThread(fIn_d, fOut_d, t, dt);
  //fi.nextTime();
  fi.update();
}


bulk::bulk(float* f0_h, float t0, float dt, int ntot){
  _ntot = _ntot;
  _nbytes = ntot*sizeof(int);
  CUDA_STATUS(hipMalloc((void**) &_fIn_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &_fOut_d, _nbytes));
  CUDA_STATUS(hipMemcpy(_fIn_d, f0_h, _nbytes, hipMemcpyHostToDevice));
}

bulk::~bulk(){
  hipFree((void*) _fIn_d);
  hipFree((void*) _fOut_d);
}

void bulk::nextTime(){
  fGrid<<<1, _ntot>>>(_fIn_d, _fOut_d, _t, _dt);
  hipDeviceSynchronize();
}

void bulk::output(float* f_h){
  cout << "In output: " << endl;
  cout << f_h[5] << endl;
  CUDA_STATUS(hipMemcpy(f_h, _fOut_d, _nbytes, hipMemcpyDeviceToHost));
}
