#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "bulk.cuh"
#include "fThread.cuh"

#define CUDA_STATUS(value){\
  hipError_t _m_cudaStat = value;\
  if ( _m_cudaStat != hipSuccess ){\
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
  }\
}

__global__ void fGrid(float* fIn_d, float *fOut_d, float t, float dt, int ntot){
  fThread fi = fThread(fIn_d, fOut_d, t, dt);
  //fi.print();
  fi.setntot(ntot);
  fi.nextTime();
  fi.update();
}


bulk::bulk(float* f0_h, float t0, float dt, int ntot){
  _ntot = ntot; _t = t0; _dt = dt;
  _nbytes = _ntot*sizeof(int);
  //cout << _ntot << ", " <<_nbytes << endl;
  CUDA_STATUS(hipMalloc((void**) &_fIn_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &_fOut_d, _nbytes));
  CUDA_STATUS(hipMemcpy(_fIn_d, f0_h, _nbytes, hipMemcpyHostToDevice));
}

bulk::~bulk(){
  hipFree((void*) _fIn_d);
  hipFree((void*) _fOut_d);
}

void bulk::nextTime(){
  fGrid<<<_ntot, 1>>>(_fIn_d, _fOut_d, _t, _dt, _ntot);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess){
    printf("Kernel call in bulk::nextTime:\n");
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(err), __LINE__, __FILE__);
    exit(1);
  }
  
  CUDA_STATUS(hipDeviceSynchronize());
  _t += _dt;
}

void bulk::output(float* f_h){
  CUDA_STATUS(hipMemcpy(f_h, _fOut_d, _nbytes, hipMemcpyDeviceToHost));
}
