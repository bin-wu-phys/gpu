#include <iostream>
#include <cmath>
using namespace std;

#include "Lattice.cuh"
#define PI 3.14159265
#include "macros.h"

Lattice::Lattice(int nr, int nphit, int npt, int nvzt, float rMax = 3.0f, float pMax = 3.0f, float vMax = 10.0f){
  _nr = nr; _nphit = nphit; _npt = npt; _nvzt = nvzt;
  _rMax = rMax; _pMax = pMax; _vMax = vMax;
  
  _r = new float[_nr]; _phit = new float[_nphit];
  _pt = new float[_npt]; _vzt = new float[_nvzt];

  calc();
}

void Lattice::calc(){
  float dl = _rMax/((float)(_nr - 1.0));

  for(int i=0; i<_nr; i++){
    _r[i] = dl*i;
    //cout << _r[i] << " ";
  }

  //cout << endl;

  dl = PI/((float)(_nphit - 1.0));

  for(int i=0; i<_nphit; i++){
    _phit[i] = dl*i;
    //cout << _phit[i] << " ";
  }

  //cout << endl;

  dl = _pMax/((float)(_npt - 1.0));

  for(int i=0; i<_npt; i++){
    _pt[i] = dl*i;
    //cout << _pt[i] << " ";
  }

  //cout << endl;

  dl = _vMax/((float)(_nvzt - 1.0));

  for(int i=0; i<_nvzt; i++){
    _vzt[i] = expl(dl*i);
    _vzt[i] = (_vzt[i] - 1.0)/(_vzt[i] + 1.0);
    //cout << _vzt[i] << " ";
  }

}

Lattice::~Lattice(){
  //The standard form of operator delete[] will take only one parameter.
  delete [] _r; delete [] _phit; delete [] _pt; delete [] _vzt;
}

void Lattice::toConstMem(const void* rc, const void* phitc, const void* ptc, const void* vztc){
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(rc), _r, _nr*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(phitc), _phit, _nphit*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(ptc), _pt, _npt*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(vztc), _vzt, _nvzt*sizeof(float)));
}

