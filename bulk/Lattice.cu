#include <iostream>
#include <cmath>
using namespace std;

#include "Lattice.cuh"
#include "macros.h"

Lattice::Lattice(int nr, int nphit, int npt, int nvzt, float rMax, float pMax, float vMax){
  _nr = nr; _nphit = nphit; _npt = npt; _nvzt = nvzt;
  _rMax = rMax; _pMax = pMax; _vMax = vMax;
  
  _r = new float[_nr]; _phit = new float[_nphit];
  _pt = new float[_npt]; _vzt = new float[_nvzt];

  calc();
}

void Lattice::set_t0(float t0){
  _t0 = t0;
}

float Lattice::get_t0(){
  return _t0;
}

void Lattice::calc(){
  float dl = _rMax/((float)(_nr - 1.0));

  for(int i=1; i<=_nr; i++){
    _r[i] = dl*i;
    //cout << _r[i] << " ";
  }

  //cout << endl;

  dl = PI/((float)(_nphit - 1.0));

  for(int i=0; i<_nphit; i++){
    _phit[i] = dl*i;
    //cout << _phit[i] << " ";
  }

  //cout << endl;

  dl = _pMax/((float)(_npt - 1.0));

  for(int i=0; i<_npt; i++){
    _pt[i] = dl*i;
    //cout << _pt[i] << " ";
  }

  //cout << endl;

  dl = _vMax/((float)(_nvzt - 1.0));

  for(int i=0; i<_nvzt; i++){
    _vzt[i] = expl(dl*i);
    _vzt[i] = (_vzt[i] - 1.0)/(_vzt[i] + 1.0);
    //cout << _vzt[i] << " ";
  }

}

Lattice::~Lattice(){
  //The standard form of operator delete[] will take only one parameter.
  delete [] _r; delete [] _phit; delete [] _pt; delete [] _vzt;
}

int Lattice::get_nr(){
  return _nr;
}

int Lattice::get_nphit(){
  return _nphit;
}

int Lattice::get_npt(){
  return _npt;
}

int Lattice::get_nvzt(){
  return _nvzt;
}

float Lattice::get_r(int i){
  if(i>=0 && i<_nr){
    return _r[i];
  }
  else{
    cout << "Error at line " << __LINE__ << " in file " << __FILE__ << ":\n";
    cout << "The index of Lattice::_r has to be in the range [0, " << _nr -1 << "]." << endl;
    exit(EXIT_FAILURE);
  }
}

float Lattice::get_phit(int i){
  if(i>=0 && i<_nphit){
    return _phit[i];
  }
  else{
    cout << "Error at line " << __LINE__ << " in file " << __FILE__ << ":\n";
    cout << "The index of Lattice::_phit has to be in the range [0, " << _nphit -1 << "]." << endl;
    exit(EXIT_FAILURE);
  }
}

float Lattice::get_pt(int i){
  if(i>=0 && i<_npt){
    return _pt[i];
  }
  else{
    cout << "Error at line " << __LINE__ << " in file " << __FILE__ << ":\n";
    cout << "The index of Lattice::_pt has to be in the range [0, " << _npt -1 << "]." << endl;
    exit(EXIT_FAILURE);
  }
}

float Lattice::get_vzt(int i){
  if(i>=0 && i<_nvzt){
    return _vzt[i];
  }
  else{
    cout << "Error at line " << __LINE__ << " in file " << __FILE__ << ":\n";
    cout << "The index of Lattice::_vzt has to be in the range [0, " << _nvzt -1 << "]." << endl;
    exit(EXIT_FAILURE);
  }
}


void Lattice::toConstMem(const void* rc, const void* phitc, const void* ptc, const void* vztc){
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(rc), _r, _nr*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(phitc), _phit, _nphit*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(ptc), _pt, _npt*sizeof(float)));
  CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(vztc), _vzt, _nvzt*sizeof(float)));
}
