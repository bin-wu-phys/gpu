#include "hip/hip_runtime.h"
#include "Coordinates.cuh"


__device__ __host__ Coordinates::Coordinates(float t0){
  _t0 = t0; _t = _t0;
}

__device__ __host__ Coordinates::Coordinates(float t0, float t){
  _t0 = t0; _t = t;
}

__device__ __host__ void Coordinates::set_t(float t){
  _t = t;
}

__device__ __host__ void Coordinates::calc(float r, float phirt, float pt, float vzt){
  float vzt2 = vzt*vzt;
  float pcon = sqrtf(vzt2 + (1.0 - vzt2)*_t*_t/(_t0*_t0));
  
  _tg = tanf(0.5*phirt)*expf(-_t0*(pcon - 1.0)/(r*sqrtf(1.0-vzt2)));
  _p = pt*pcon; _vz = vzt*_t0/(_t*pcon);
}

__device__ __host__ float Coordinates::cos_phir(){
  return (1.0-_tg*_tg)/(1.0+_tg*_tg);
}

__device__ __host__ float Coordinates::sin_phir(){
  return 2.0*_tg/(1.0+_tg*_tg);
}

