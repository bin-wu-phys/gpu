#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "KinTran.cuh"
#include "fThread.cuh"
#include "macros.h"

__global__ void fGrid(float* fIn_d, float *fOut_d, float t, float dt, KernelParas *kps_all){
  KernelParas kps;
  kps._f_r = fIn_d + blockIdx.x*kps_all->_nphit*kps_all->_npt*kps_all->_nvzt;
  kps._nphit = kps_all->_nphit; kps._npt = kps_all->_npt; kps._nvzt = kps_all->_nvzt;
  fThread fi = fThread(fIn_d, fOut_d, t, dt, &kps);
  //fi.print();
  fi.nextTime();
  fi.update();
}


KinTran::KinTran(InitCond* init, float dt){
  _latt = init-> _latt;
  _ntot = _latt->get_nr()*_latt->get_nphit()*_latt->get_npt()*_latt->get_nvzt();
  _nbytes = _ntot*sizeof(float);
  _t = init->get_tInit(); _dt = dt;
  //cout << _ntot << ", " <<_nbytes << endl;
  CUDA_STATUS(hipMalloc((void**) &_f, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &_fPre, _nbytes));
  init->toGlobalMem(_fPre);
}

KinTran::~KinTran(){
  hipFree((void*) _f);
  hipFree((void*) _fPre);
}

void KinTran::nextTime(){
  KernelParas kps;
  kps._f_r = _f; kps._nphit = _latt->get_nphit();
  kps._npt = _latt->get_npt(); kps._nvzt = _latt->get_nvzt();
  
  fGrid<<<_latt->get_nr(), dim3(_latt->get_nphit(), _latt->get_npt(), _latt->get_nvzt())>>>(_f, _fPre, _t, _dt, &kps);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess){
    printf("Kernel call in KinTran::nextTime:\n");
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(err), __LINE__, __FILE__);
    exit(1);
  }
  
  CUDA_STATUS(hipDeviceSynchronize());
  _t += _dt;
}

void KinTran::output(float* f_h){
  CUDA_STATUS(hipMemcpy(f_h, _f, _nbytes, hipMemcpyDeviceToHost));
}
