#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "KinTran.cuh"
#include "fThread.cuh"
#include "macros.h"

__global__ void fGrid(float* fIn_d, float *fOut_d, float t, float dt, int ntot){
  fThread fi = fThread(fIn_d, fOut_d, t, dt);
  //fi.print();
  fi.setntot(ntot);
  fi.nextTime();
  fi.update();
}


KinTran::KinTran(float* f0_h, float t0, float dt, int nx, int nphi, int npT, int npz){
  _nx = nx; _nphi = nphi; _npT = npT; _npz = npz;
  _ntot = _nx*_nphi*_npT*_npz;
  _nbytes = _ntot*sizeof(float);
  _t = t0; _dt = dt;
  //cout << _ntot << ", " <<_nbytes << endl;
  CUDA_STATUS(hipMalloc((void**) &_fIn_d, _nbytes));
  CUDA_STATUS(hipMalloc((void**) &_fOut_d, _nbytes));
  CUDA_STATUS(hipMemcpy(_fIn_d, f0_h, _nbytes, hipMemcpyHostToDevice));
}

KinTran::~KinTran(){
  hipFree((void*) _fIn_d);
  hipFree((void*) _fOut_d);
}

void KinTran::nextTime(){
  int nt = 32*4;
  fGrid<<<(_ntot+nt-1)/nt, nt>>>(_fIn_d, _fOut_d, _t, _dt, _ntot);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess){
    printf("Kernel call in KinTran::nextTime:\n");
    fprintf( stderr , "Error %s at line %d in file %s \n", hipGetErrorString(err), __LINE__, __FILE__);
    exit(1);
  }
  
  CUDA_STATUS(hipDeviceSynchronize());
  _t += _dt;
}

void KinTran::output(float* f_h){
  CUDA_STATUS(hipMemcpy(f_h, _fOut_d, _nbytes, hipMemcpyDeviceToHost));
}
