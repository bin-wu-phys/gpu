#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include "Coordinates.cuh"

__global__ void test_coor(float r, float phirt, float pt, float vzt){
  Coordinates coor(0.1, 1.0);
  coor.calc(r, phirt, pt, vzt);

  printf("%f on (%d, %d).\n", coor.cos_phir(), blockIdx.x, threadIdx.x);

}

int main(){
  Coordinates coor(0.1, 1.0);
  float r = 1.0, phirt = 0.1, pt = 1.0, vzt = 0.3;
  coor.calc(r, phirt, pt, vzt);

  printf("%f on the host.\n", coor.cos_phir());

  test_coor<<<2, 3>>>(r, phirt, pt, vzt);

  hipDeviceSynchronize();
  return 0;
}
