#include <iostream>
using namespace std;

#include "GPU.h"

int GPU::CoresPerSM(int major, int minor){
  int cores = 0;
  switch (major){
     case 2: // Fermi
      if (minor == 1) cores = 48;
      else cores = 32;
      break;
    case 3: // Kepler
      cores = 192;
      break;
    case 5: // Maxwell
      cores = 128;
      break;
    case 6: // Pascal
      if ((minor == 1) || (minor == 2)) cores = 128;
      else if (minor == 0) cores = 64;
      else printf("Unknown device type\n");
      break;
    case 7: // Volta and Turing
      if ((minor == 0) || (minor == 5)) cores = 64;
      else printf("Unknown device type\n");
      break;
    case 8: // Ampere
      if (minor == 0) cores = 64;
      else if (minor == 6) cores = 128;
      else printf("Unknown device type\n");
      break;
    default:
      printf("Unknown device type\n"); 
      break;
  } 
  return cores;
}

void GPU::info(){
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cout << "Device Number: " <<  i << endl;
    cout << "  Device name: " << prop.name << endl;
    cout << "  Compute capability: " << prop.major << "." << prop.minor << endl;
    int cs = CoresPerSM(prop.major, prop.minor);
    cout << "  (SMs, Cuda cores/SM, Cuda cores): (" <<  prop.multiProcessorCount;
    cout << ", " << cs << ", " << cs* prop.multiProcessorCount << ")" << endl;
    cout << "  Maximum number of threads per block: " << prop.maxThreadsPerBlock << endl;
    cout << "  Warp Size: " << prop.warpSize << endl;
    cout << "  Maximum size of a block: (" << prop.maxThreadsDim[0] << ", ";
    cout << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")" << endl;
    cout << "  Maximum size of a grid: (" << prop.maxGridSize[0] << ", ";
    cout << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << endl;
    cout << "  Shared memory available per multiprocess: " << prop.sharedMemPerMultiprocessor/1024.0 << " kb" << endl;
    cout << "  Shared memory available per block: " << prop.sharedMemPerBlock/1024.0 << " kb" << endl;
    cout << "  Total number of registers available per block: " << prop.regsPerBlock << endl;
    cout << "  Total memory: " << prop.totalGlobalMem/1073741824 << " Gb" << endl;
    cout << "  Peak Memory Bandwidth " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << " Gb/s" << endl;
  }
}
