#include <iostream>
using namespace std;

#include "GPU.h"

int GPU::CoresPerSM(int major, int minor){
  int cores = 0;
  switch (major){
     case 2: // Fermi
      if (minor == 1) cores = 48;
      else cores = 32;
      break;
    case 3: // Kepler
      cores = 192;
      break;
    case 5: // Maxwell
      cores = 128;
      break;
    case 6: // Pascal
      if ((minor == 1) || (minor == 2)) cores = 128;
      else if (minor == 0) cores = 64;
      else printf("Unknown device type\n");
      break;
    case 7: // Volta and Turing
      if ((minor == 0) || (minor == 5)) cores = 64;
      else printf("Unknown device type\n");
      break;
    case 8: // Ampere
      if (minor == 0) cores = 64;
      else if (minor == 6) cores = 128;
      else printf("Unknown device type\n");
      break;
    default:
      printf("Unknown device type\n"); 
      break;
  } 
  return cores;
}

void GPU::info(){
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cout << "Device Number: " <<  i << endl;
    cout << "  Device name: " << prop.name << endl;
    cout << "  Compute capability: " << prop.major << "." << prop.minor << endl;
    int cs = CoresPerSM(prop.major, prop.minor);
    cout << "  (SMs, Cuda cores/SM, Cuda cores): (" <<  prop.multiProcessorCount;
    cout << ", " << cs << ", " << cs* prop.multiProcessorCount << ")\n" << endl;
    
    cout << "  Warp Size: " << prop.warpSize << endl;
    cout << "  Maximum number of threads per block: " << prop.maxThreadsPerBlock << endl;
    cout << "  Maximum size of a block: (" << prop.maxThreadsDim[0] << ", ";
    cout << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")" << endl;
    cout << "  Maximum size of a grid: (" << prop.maxGridSize[0] << ", ";
    cout << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << endl;
    cout << "  Maximum number of threads per SM: " << prop.maxThreadsPerMultiProcessor << endl;
    cout << "  Maximum number of warps per SM: " << prop.maxThreadsPerMultiProcessor/prop.warpSize << '\n' << endl;    
    
    cout << "  Shared memory available per multiprocess: " << prop.sharedMemPerMultiprocessor/1024.0 << " kb" << endl;
    cout << "  Shared memory available per block: " << prop.sharedMemPerBlock/1024.0 << " kb" << endl;
    cout << "  Total number of registers available per block: " << prop.regsPerBlock << endl;
    cout << "  Total number of registers available per SM: " << prop.regsPerMultiprocessor << endl;
    cout << "  Total memory: " << prop.totalGlobalMem/1073741824 << " Gb" << endl;
    cout << "  Peak Memory Bandwidth " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << " Gb/s" << endl;
    cout << "\nCUDA:\n" << endl;
    int dv, rv;
    hipDriverGetVersion(&dv);
    hipRuntimeGetVersion(&rv);
    cout << "   CUDA Driver Version: " << dv/1000 << ", " << (dv%100/10) << endl;   cout << "   CUDA Runtime Version: " << rv/1000 << ", " << (rv%100/10) << endl;
  }
}
